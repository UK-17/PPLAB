#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
__global__ void revs(char * d_res , char * d_buffer , int * d_length)
{
    int threadid = threadIdx.x;
    int length = *d_length ;
    d_res[threadid]=d_buffer[length-1-threadid];
    
 }
int main(void)
{

char buffer[100];
char res[100];

printf("\nEnter string :");
gets(buffer);

int length = strlen(buffer) ;

char * d_res ; 
char * d_buffer ; 
int * d_length;

hipMalloc((void **)&d_res,(length+1) * sizeof(char));
hipMalloc((void **)&d_buffer,(length+1)*sizeof(char));
 hipMalloc((void **)&d_length,sizeof(int));

hipMemcpy(d_buffer,buffer,(length+1)*sizeof(char),hipMemcpyHostToDevice);
 hipMemcpy(d_length,&length,sizeof(int),hipMemcpyHostToDevice);


revs<<<1,length>>>(d_res,d_buffer,d_length);
hipError_t error = hipGetLastError();
 if(error!= hipSuccess)
 {
	 	printf("%s\n",hipGetErrorString(error));
 }
hipMemcpy(&res,d_res,sizeof(char) * length,hipMemcpyDeviceToHost);
res[length]='\0';
printf("Result :%s\n",res);

}
