#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<string.h>
__global__ void repeat(char * d_resbuffer , char * d_buffer, int * d_length)
{
		
	int id = threadIdx.x ;
	int start = id * (*d_length);
	for(int i = 0 ; i<=(*d_length)-1;i++)
	{
		d_resbuffer[start] = d_buffer[i];
		start ++ ;
	}
}
int main(void)
{

char buffer[100];
printf("\nEnter string :");
scanf("%s",buffer);

char res[100] = "";

int length = strlen(buffer) ;

int n;
printf("Enter n=?: ");
scanf("%d",&n);

 
char * d_resbuffer ; 
  
char * d_buffer ; 
int * d_length ;

hipMalloc((void **)&d_resbuffer,(length*n+1) * sizeof(char));
hipMalloc((void **)&d_buffer,(length+1)*sizeof(char));
hipMalloc((void **)&d_length , sizeof(int));
  
hipMemcpy(d_buffer,buffer,(length+1)*sizeof(char),hipMemcpyHostToDevice);
hipMemcpy(d_length , &length , sizeof(int),hipMemcpyHostToDevice);


repeat<<<1,n>>>(d_resbuffer,d_buffer,d_length);
hipError_t error = hipGetLastError();
 if(error!= hipSuccess)
 {
	 	printf("%s\n",hipGetErrorString(error));
 }
hipMemcpy(res,d_resbuffer,(length*n)*sizeof(char),hipMemcpyDeviceToHost);

 int location = length * n ;
 res[location]='\0';
 
printf("Result :%s\n",res);

}

