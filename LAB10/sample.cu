#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<unistd.h>
#define BLOCK_WIDTH 2
#define TILE_WIDTH 2
#define WIDTH 4

__device__ int getTid(){
	int blockSkip=(blockIdx.y*gridDim.x*blockDim.x*blockDim.y);
	int rowSkip=(threadIdx.y*gridDim.x*blockDim.x);
	int rowDisplacement=(blockIdx.x*blockDim.x)+threadIdx.x;
	int tid=blockSkip+rowSkip+rowDisplacement;
	return tid;
}

__global__ void MatAddElement(int *a,int *b,int *c){
	int tid=getTid();
	c[tid]=a[tid]+b[tid];
}

int main()
{
	int *matA,*matB,*matSum;
	int *da,*db,*dc;
	int m,n;
	m=n=WIDTH;
	int size=sizeof(int)*WIDTH*WIDTH;
	printf("Enter the matrix A elements(4x4):\n");
	matA=(int*)malloc(size);
	for(int i=0;i<WIDTH*WIDTH;i++){
		scanf("%d",&matA[i]);
	}
	printf("Enter the matrix B elements(4x4):\n");
	matB=(int*)malloc(size);
	for(int i=0;i<WIDTH*WIDTH;i++){
		scanf("%d",&matB[i]);
	}
	matSum=(int*)malloc(size);
	hipMalloc((void**)&da,size);
	hipMalloc((void**)&db,size);
	hipMalloc((void**)&dc,size);
	hipMemcpy(da,matA,size,hipMemcpyHostToDevice);
	hipMemcpy(db,matB,size,hipMemcpyHostToDevice);
	int NumBlocks=WIDTH/BLOCK_WIDTH;
	dim3 grid (NumBlocks,NumBlocks);
	dim3 block (BLOCK_WIDTH,BLOCK_WIDTH);
	MatAddElement<<<grid,block>>>(da,db,dc);
	hipMemcpy(matSum,dc,size,hipMemcpyDeviceToHost);
	printf("Result :\n");
	for(int i=0;i<m;i++){
		for(int j=0;j<n;j++){
			printf("%d\t",matSum[i*n+j]);
		}
		printf("\n");
	}
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	free(matA);
	free(matB);
	free(matSum);
	return 0;
}
