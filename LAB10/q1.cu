#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<unistd.h>
#define BLOCK_WIDTH 2
#define TILE_WIDTH 2
#define WIDTH 4

__device__ int getTid()
{
	int blockSkip=(blockIdx.y*gridDim.x*blockDim.x*blockDim.y);
	int rowSkip=(threadIdx.y*gridDim.x*blockDim.x);
	int rowDisplacement=(blockIdx.x*blockDim.x)+threadIdx.x;
	int tid=blockSkip+rowSkip+rowDisplacement;
	return tid;
}

__global__ void MatMulElement(int *a,int *b,int *c)
{
	int tid=getTid();
	int rId=tid%4;
	int cId=tid-(tid%4);
	int sum=0;
	for(int i=0;i<4;i++)
		sum+=a[cId+i]*b[rId+4*i];
	
	c[tid]=sum;
}


int main()
{
	int *matA,*matB,*res_matrix;
	int *da,*db,*dc;
	int m,n;
	
	m=n=WIDTH;
	int size=sizeof(int)*WIDTH*WIDTH;
	
	printf("Enter the matrix A elements(4x4):\n");
	matA=(int*)malloc(size);
	for(int i=0;i<WIDTH*WIDTH;i++)
		scanf("%d",&matA[i]);
	
	printf("Enter the matrix B elements(4x4):\n");
	matB=(int*)malloc(size);
	for(int i=0;i<WIDTH*WIDTH;i++)
		scanf("%d",&matB[i]);
	
	res_matrix=(int*)malloc(size);
	
	hipMalloc((void**)&da,size);
	hipMalloc((void**)&db,size);
	hipMalloc((void**)&dc,size);
	
	hipMemcpy(da,matA,size,hipMemcpyHostToDevice);
	hipMemcpy(db,matB,size,hipMemcpyHostToDevice);
	
	int NumBlocks=WIDTH/BLOCK_WIDTH;
	dim3 grid (NumBlocks,NumBlocks);
	dim3 block (BLOCK_WIDTH,BLOCK_WIDTH);
	
	MatMulElement<<<grid,block>>>(da,db,dc);
	
	hipMemcpy(res_matrix,dc,size,hipMemcpyDeviceToHost);
	
	printf("Result :\n");
	for(int i=0;i<m;i++)
	{
		for(int j=0;j<n;j++)
			printf("%d\t",res_matrix[i*n+j]);
		printf("\n");
	}	
		printf("\n");
	
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	
	free(matA);
	free(matB);
	free(res_matrix);
	
	return 0;
}
