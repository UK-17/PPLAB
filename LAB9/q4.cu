#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void kernel(int *a,int *c,int m,int n){
	int id1=threadIdx.x;
	int id2=blockIdx.x;
	int pos=id2*n+id1;
	if((id2==0||id2==m-1)||(id1==0||id1==n-1)) c[pos]=a[pos];
	else{
		int num=a[pos];int f=1;
		int co=0;
		do{
			int d=num%2;
			co=co+(1-d)*f;
			f=f*10;
			num=num/2;
		}while(num!=0);
		c[pos]=co;
	}
}

int main(){
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;
	printf("Enter N: ");
	int n;scanf("%d",&n);
	printf("Enter M: ");
	int m;scanf("%d",&m);
	int size=sizeof(int)*n*m;
	a=(int *)malloc(size);
	b=(int *)malloc(size);
	c=(int *)malloc(size);

	printf("Enter the Matrix A: \n");
	for(int i=0;i<n*m;i++){
		scanf("%d",&a[i]);
	}
	printf("\n");

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	kernel<<<m,n>>>(d_a,d_c,m,n);

	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("Result Matrix is: \n");
	for(int i=0;i<m*n;i++){
		if(i%n==0) printf("\n");
		printf("%5d ",c[i]);
	}
	printf("\n");

	getchar();
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
