#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void kernel(int *a,int *b,int *c,int n){
	int id1=threadIdx.x;
	int id2=blockIdx.x;
	int pos=id2*n+id1;
	c[pos]=a[pos]+b[pos];
}

int main(){
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;
	printf("Enter the order of the matrix: ");
	int n;scanf("%d",&n);
	int size=sizeof(int)*n*n;
	a=(int *)malloc(size);
	b=(int *)malloc(size);
	c=(int *)malloc(size);

	printf("Enter the Matrix A: \n");
	for(int i=0;i<n*n;i++){
		scanf("%d",&a[i]);
	}
	printf("\n");

	printf("Enter the Matrix B: \n");
	for(int i=0;i<n*n;i++){
		scanf("%d",&b[i]);
	}
	printf("\n");

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	kernel<<<n,n>>>(d_a,d_b,d_c,n);

	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("Result Matrix is: \n");
	for(int i=0;i<n*n;i++){
		if(i%n==0) printf("\n");
		printf("%5d ",c[i]);
	}
	printf("\n");

	getchar();
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
