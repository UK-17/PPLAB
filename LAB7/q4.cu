#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void trig (float *input, float *output) {
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
    printf("idx = %d\n", idx);
    output[idx] = sin(input[idx]);
}

int main () {
    
    int M,N;
    printf("Blocks M:");
    scanf("%d",&M);
    printf("Threads N:");
    scanf("%d",&N);

    
    // Host copies of the variables
    float input[M*N], output[M*N];

    int i, j;
    printf("Enter %d elements:\n",M*N);
    for (i = 0; i < M*N; ++i) 
        scanf("%f",&input[i]);
   

    // Device copies of the variables
    float *d_a, *d_b;

    int size = sizeof(float) * M*N;

    // Allocate memories to device copies of the objects
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    

    // Copy inputs to device
    hipMemcpy(d_a, &input, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &output, size, hipMemcpyHostToDevice);

    // Launch kernel onto the device
    trig<<<M, N>>>(d_a, d_b);

    // Copy the result back to the host
    hipMemcpy(&output, d_b, size, hipMemcpyDeviceToHost);

    // Output
    printf("X     :");
    for (j = 0; j < M*N; ++j) 
        printf("\t%f", input[j]);
    
    printf("\n");

    printf("sin(x):");
    for (j = 0; j < M*N; ++j) 
        printf("\t%f", output[j]);
    
    printf("\n");


    hipFree(d_a);
    hipFree(d_b);


	getchar();

    return 0;
}
