#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void add (int *A, int *B, int *C) {
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
    printf("idx = %d\n", idx);
    C[idx] = A[idx] + B[idx];
}

int main () {
    
    int M,N;
    printf("Blocks M:");
    scanf("%d",&M);
    printf("Threads N:");
    scanf("%d",&N);

    
    // Host copies of the variables
    int A[M*N], B[M*N], C[M*N];

    int i, j;
    printf("Enter %d elements in A:\n",M*N);
    for (i = 0; i < M*N; ++i) 
        scanf("%d",&A[i]);
    printf("Enter %d elements in B:\n",M*N);    
    for (i = 0; i < M*N; ++i) 
        scanf("%d",&B[i]);
    

    // Device copies of the variables
    int *d_a, *d_b, *d_c;

    int size = sizeof(int) * M*N;

    // Allocate memories to device copies of the objects
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &B, size, hipMemcpyHostToDevice);

    // Launch kernel onto the device
    add<<<M, N>>>(d_a, d_b, d_c);

    // Copy the result back to the host
    hipMemcpy(&C, d_c, size, hipMemcpyDeviceToHost);

    // Output
    printf("A:\n");
    for (j = 0; j < M*N; ++j) {
        printf("%d\t", A[j]);
    }
    printf("\n");

    printf("B:\n");
    for (j = 0; j < M*N; ++j) {
        printf("%d\t", B[j]);
    }
    printf("\n");

    printf("A + B:\n");
    for (j = 0; j < M*N; ++j) {
        printf("%d\t", C[j]);
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

	getchar();

    return 0;
}
