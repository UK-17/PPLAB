#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void func (int *X, int *Y, int *A ) {
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
    printf("idx = %d\n", idx);
    Y[idx] = *A * X[idx] + Y[idx];
}

int main () {
    
    int M,N;
    printf("Blocks M:");
    scanf("%d",&M);
    printf("Threads N:");
    scanf("%d",&N);

    
    // Host copies of the variables
    int X[M], Y[M], A;

    int i, j;
    printf("Enter const. A : ");
    scanf("%d",&A);
    printf("Enter %d elements in X:\n",M*N);
    for (i = 0; i < M*N; ++i) 
        scanf("%d",&X[i]);
    printf("Enter %d elements in Y:\n",M*N);    
    for (i = 0; i < M*N; ++i) 
        scanf("%d",&Y[i]);

    printf("X:\n");
    for (j = 0; j < M*N; ++j) {
        printf("%d\t", X[j]);
    }
    printf("\n");

    printf("Y:\n");
    for (j = 0; j < M*N; ++j) {
        printf("%d\t", Y[j]);
    }
    printf("\n");

    

    // Device copies of the variables
    int *d_x, *d_y , *d_a;

    int size = sizeof(int) * M*N;

    // Allocate memories to device copies of the objects
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);
    hipMalloc((void**)&d_a, sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_x, &X, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, &Y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_a, &A, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel onto the device
    func<<<M, N>>>(d_x, d_y, d_a);

    // Copy the result back to the host
    hipMemcpy(&Y, d_y, size, hipMemcpyDeviceToHost);

    // Output
    printf("Y:\n");
    for (j = 0; j < M*N; ++j) {
        printf("%d\t", Y[j]);
    }
    printf("\n");
    
    

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_a);

	getchar();

    return 0;
}
